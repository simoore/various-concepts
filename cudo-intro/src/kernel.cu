#include <algorithm>
#include <cstdlib>
#include <cstdio>

#include "hip/hip_runtime.h"


__global__ void sumArrayGpu(const int *a, const int *b, int *c, int size) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size) {
        c[gid] = a[gid] + b[gid];
    }
}

void sumArrayCpu(const int *a, const int *b, int *c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
} 

int main() {
    const int arraySize = 10000;
    const int blockSize = 128;
    const int numBytes = arraySize * sizeof(int);
    hipError_t error;

    int *hA = reinterpret_cast<int *>(std::malloc(numBytes));
    int *hB = reinterpret_cast<int *>(std::malloc(numBytes));
    int *gpuResults = reinterpret_cast<int *>(std::malloc(numBytes));
    int *hC = reinterpret_cast<int *>(std::malloc(numBytes));

    auto initFunc = [&](int *arr) {
        for (int i = 0; i < arraySize; i++) {
            arr[i] = static_cast<int>(rand() & 0xFF);
        }
    };

    initFunc(hA);
    initFunc(hB);
    sumArrayCpu(hA, hB, hC, arraySize);

    int * dA, *dB, *dC;

    error = hipMalloc(&dA, numBytes);
    if (error != hipSuccess) {
        fprintf(stderr, " Error : %s %s %d\n", hipGetErrorString(error), __FILE__, __LINE__);
        return 1;
    }
    hipMalloc(&dB, numBytes);
    hipMalloc(&dC, numBytes);

    hipMemcpy(dA, hA, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, numBytes, hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid(arraySize / block.x + 1);

    sumArrayGpu<<<grid, block>>>(dA, dB, dC, arraySize);
    
    // Wait til operation has finished.
    hipDeviceSynchronize();

    hipMemcpy(gpuResults, dC, numBytes, hipMemcpyDeviceToHost);
    if (std::equal(hC, hC + arraySize, gpuResults)) {
        printf("Arrays are the same\n");
    } else {
        printf("Arrays are different\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);
    free(hC);
    free(gpuResults);

    hipDeviceReset();
    return 0;
}