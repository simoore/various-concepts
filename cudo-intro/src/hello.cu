// compile with: nvcc src/hello.cu -o hello


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void cudaHello() {
    printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d, Hello World from GPU!\n", 
        threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {

    // A block has 2 threads in each dimension
    dim3 block(2, 2, 2);

    // We are launching 4 blocks, 2 in x dimension, 2 in y dimension, 1 in z dimension
    dim3 grid(2, 2, 1);

    // We are launching 20 threads to run the cudaHello function
    cudaHello<<<grid, block>>>(); 
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}