#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        std::cout << "No CUDA supported device found" << std::endl;
    }

    int devNo = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);

    std::cout << "Device " << devNo << ": " << iProp.name << std::endl;
    std::cout << "  number of multiprocessors    : " << iProp.multiProcessorCount << std::endl;
    std::cout << "  clock rate                   : " << iProp.clockRate << std::endl;
    std::cout << "  compute capability           : " << iProp.major << "." << iProp.minor << std::endl;
    std::cout << "  total global memory          : " << (iProp.totalGlobalMem / 1024.0) << std::endl;
    std::cout << "  total constant memory        : " << (iProp.totalConstMem / 1024.0) << std::endl;
    std::cout << "  total shared memory per block: " << (iProp.sharedMemPerBlock / 1024.0) << std::endl;
    std::cout << "  total shared memory per MP   : " << (iProp.sharedMemPerMultiprocessor / 1024.0) << std::endl;

    hipDeviceReset();
    return 0;
}